#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
#include <array>
#include <math.h>
#include <chrono>     
#include <iomanip>
using namespace std;
using namespace std::chrono; 

int n;
int num_cluster;
int dims;
string inputfilename;
int max_num_iter;
float threshold;
bool flag;
unsigned int seed = 8675309;

static unsigned long int nextstd = 1;
static unsigned long kmeans_rmax = 32767;
int kmeans_rand() {
    nextstd = nextstd * 1103515245 + 12345;
    return (unsigned int)(nextstd/65536) % (kmeans_rmax+1);
}
void kmeans_srand(unsigned int funcseed) {
    nextstd = funcseed;
}
__global__ void findNearestCentroids(int* labels, float* dataset, float* centroids, int* n, int* dims, int* num_cluster){
    //for(int i = 0; i < n;i++){
		extern __shared__ int s[];
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int si = threadIdx.x;
		s[si] = labels[i];
		if(i >= *n){
			return;
		}
		labels[i] = 0;
        int label = 0;
        float dist=0;
        float running = 0;
        for(int k = 0; k < *dims; k++){
            running += pow(dataset[i*(*dims)+k]-centroids[0*(*dims)+k],2);
        }
        running = sqrt(running);
        dist = running;
        for(int j = 1; j < *num_cluster; j++){
            float inner = 0;
            for(int k = 0; k < *dims; k++){
                inner += pow(dataset[i*(*dims)+k]-centroids[j*(*dims)+k],2);
            }
            inner = sqrt(inner);
            if(inner < dist){
               label = j;
               dist = inner;
            }
        }
		s[si] = label;
		labels[i] = s[si];

}
void randomCentroids(float centroids[],float dataset[]){
	int count = 0;
    for (int i=0; i<num_cluster; i++){
        int index = kmeans_rand() % n;
        for (int j = 0; j < dims; j++){
			centroids[count] = dataset[index*dims+j];
			count++;
            //centroids.push_back(dataset[index*dims+j]);
        }
    }
}
void zeroArray(float centroids[]){
	for(int i = 0; i < num_cluster*dims;i++){
		centroids[i] = 0;
	}
}
void averageLabeledCentroids(float newcentroids[], float dataset[], int labels[]){
	float labelcounts[num_cluster]={0};
   for(int i = 0; i < n; i++){
       labelcounts[labels[i]] = labelcounts[labels[i]]+1;
       for(int j = 0; j < dims; j++){
        newcentroids[labels[i]*dims+j] += dataset[i*dims+j];
       }
   }

   for(int i = 0; i < num_cluster; i++){
       for(int j = 0; j < dims; j++){
           newcentroids[i*dims+j] = newcentroids[i*dims+j]/labelcounts[i];
       }
   }
}
void converged(float oldCentroids[], float newCentroids[], int* convergedFlag){
	for(int i = 0; i < num_cluster*dims; i++){
		if(!(abs(oldCentroids[i] - newCentroids[i]) < threshold)){
			*convergedFlag = *convergedFlag + 1;
		}
	}
}
void kmeans(float dataset[]){
	float centroids[num_cluster*dims]={0};
	int labels[n]={0};
    randomCentroids(centroids, dataset);
    int iterations = 0;
    bool done = false;
	float totalduration = 0;
	//
	//DEVICE ARRAYS
	//
	float *d_centroids;
	float* d_dataset;
	int *d_labels;
	int csize = num_cluster*dims*sizeof(float);
	int dsize = n*dims*sizeof(float);
	int lsize = n*sizeof(int);
	hipMalloc((void **)&d_centroids, csize);
	hipMalloc((void **)&d_dataset, dsize);
	hipMalloc((void **)&d_labels, lsize);
	//
	//DEVICE VARIABLES
	//
	int* d_n;
	int* d_dims;
	int* d_num_cluster;
	hipMalloc((void **)&d_n, sizeof(int));
	hipMalloc((void **)&d_dims, sizeof(int));
	hipMalloc((void **)&d_num_cluster, sizeof(int));
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dims, &dims, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_num_cluster, &num_cluster, sizeof(int), hipMemcpyHostToDevice);
    while(!done){
		auto start = high_resolution_clock::now();
		float oldCentroids[num_cluster*dims]={0};
        //vector<float> oldCentroids;
        for(int i = 0; i < num_cluster*dims; i++){
			oldCentroids[i] = centroids[i];
        }
        iterations++;
		//
		//FIND NEAREST CENTROIDS
		//
		hipMemcpy(d_centroids, centroids, csize, hipMemcpyHostToDevice);
		hipMemcpy(d_labels, labels, lsize, hipMemcpyHostToDevice);
		hipMemcpy(d_dataset, dataset, dsize, hipMemcpyHostToDevice);
		int blockSize;
		int minGridSize;
		int gridSize;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*) findNearestCentroids, 0, n);
		gridSize = (n+blockSize-1)/blockSize;
		findNearestCentroids<<<gridSize, blockSize, blockSize*sizeof(int)>>>(d_labels,d_dataset,d_centroids, d_n, d_dims, d_num_cluster);
		hipDeviceSynchronize();
		hipMemcpy(labels, d_labels, lsize, hipMemcpyDeviceToHost);
		//
		//AVERAGE LABELED CENTROIDS
		//
		zeroArray(centroids);
        averageLabeledCentroids(centroids,dataset,labels);
		int convergedFlag = 0;
		converged(centroids,oldCentroids, &convergedFlag);
        done = (iterations == max_num_iter || convergedFlag==0);
		auto stop = high_resolution_clock::now();
		float duration = duration_cast<microseconds>(stop - start).count(); 
		totalduration += duration;
    }
	hipFree(d_centroids); hipFree(d_labels); hipFree(d_dataset);
	hipFree(d_n); hipFree(d_dims); hipFree(d_num_cluster);
	float time_per_iter_in_ms = totalduration/iterations;
	auto iter_to_converge = iterations;
	printf("%d,%lf\n", iter_to_converge, time_per_iter_in_ms);
	if(flag){
		for(int i = 0; i < num_cluster; i++){
			printf("%d ", i);
			for(int j = 0; j < dims; j++){
				printf("%lf ", centroids[i*dims +j]);
			}
			printf("\n");
		}
	}
	else{
		printf("clusters:");
		for (int p=0; p < n; p++)
			printf(" %d", labels[p]);		
	}
}


int main(int argc, char* argv[]){
    string inputPath;
    for(int i = 1; i < argc; i++){
        string arg = string(argv[i]);
        if(arg == "-k"){
            num_cluster = stoi(argv[++i]);
        }
        if(arg == "-d"){
            dims = stoi(argv[++i]);
        }
        if(arg == "-i"){
            inputfilename = argv[++i]; 
        }
        if(arg == "-m"){
            max_num_iter = stoi(argv[++i]);
        }
        if(arg == "-t"){
            threshold = stod(argv[++i]);
        }
        if(arg == "-c"){
            flag = true;
        }
        if(arg == "-s"){
            seed = stoi(argv[++i]);
            kmeans_srand(seed);
        }
    }
    ifstream inputfile;
    inputfile.open(inputfilename);
    string stringn;
    getline(inputfile, stringn);
    n = stoi(stringn);
	float dataset[n*dims]={0};
	int count = 0;
    for(int i = 0; i < n; i++){
        string line;
        getline(inputfile,line);
        string value="";
        bool index = false;
        for(int j = 0; j < line.size(); j++){
            if(line[j] == ' '){
                if(index == false){
                    index = true;
                    value = "";
                    continue;
                }
				dataset[count] = stod(value);
               //dataset.push_back(stod(value)); 
			   count++;
               value="";
            }
            else{
                value = value + line[j];
            }
        }
        if(value != ""){
			dataset[count] = stod(value);
			count++;
            //dataset.push_back(stod(value));
        }
    }
    kmeans(dataset);
}
