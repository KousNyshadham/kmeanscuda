#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/for_each.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

#include <string>
#include <fstream>
#include <iostream>
#include <vector>
#include <math.h>
#include <chrono>     
#include <iomanip>
using namespace std;
using namespace std::chrono; 

int n;
int num_cluster;
int dims;
string inputfilename;
int max_num_iter;
double threshold;
bool flag;
unsigned int seed = 8675309;

static unsigned long int nextstd = 1;
static unsigned long kmeans_rmax = 32767;
int kmeans_rand() {
    nextstd = nextstd * 1103515245 + 12345;
    return (unsigned int)(nextstd/65536) % (kmeans_rmax+1);
}
void kmeans_srand(unsigned int funcseed) {
    nextstd = funcseed;
}

thrust::host_vector<float> hostdataset(0);
thrust::host_vector<float> hostcentroids(0);
thrust::host_vector<int> hostlabels(0);

struct findNearestCentroids{
	int dims;
	int num_cluster;
	float* dataset;
	float* centroids;
	int * labels;

	findNearestCentroids(int _dims, int _num_cluster, float* _dataset, float* _centroids, int* _labels):
	dims(_dims), num_cluster(_num_cluster), dataset(_dataset), centroids(_centroids), labels(_labels){};

	__host__ __device__
	void operator()(int i){
        int label = 0;
        float dist=0;
        float running = 0;
        for(int k = 0; k < dims; k++){
            running += pow(dataset[i*dims+k]-centroids[0*dims+k],2);
        }
        running = sqrt(running);
        dist = running;
        for(int j = 1; j < num_cluster; j++){
            float inner = 0;
            for(int k = 0; k < dims; k++){
                inner += pow(dataset[i*dims+k]-centroids[j*dims+k],2);
            }
            inner = sqrt(inner);
            if(inner < dist){
               label = j;
               dist = inner;
            }
        }
		labels[i] = label;
	}
};

struct converged{
	double threshold;
	float* oldCentroids;
	float* newCentroids;
	int* convergedVector;

	converged(double _threshold, float* _oldCentroids, float* _newCentroids, int* _convergedVector):
	threshold(_threshold), oldCentroids(_oldCentroids), newCentroids(_newCentroids), convergedVector(_convergedVector){};


	
	__host__ __device__
	void operator()(int i){
		if(!(abs(oldCentroids[i] - newCentroids[i]) < threshold)){
			convergedVector[i] = 1;
		}
		else{
			convergedVector[i] = 0;
		}
	}
};


void randomCentroids(){
	int counter = 0;
    for (int i=0; i<num_cluster; i++){
        int index = kmeans_rand() % n;
        for (int j = 0; j < dims; j++){
			hostcentroids[counter] = hostdataset[index*dims+j];
			counter++;
        }
    }
}

struct newadd{
	int dims;
	int* labels;
	float* centroids;
	float* dataset;

	newadd(int _dims , float* _centroids, float* _dataset, int* _labels):	
	centroids(_centroids), dataset(_dataset), dims(_dims), labels(_labels){};

	__device__
	void operator()(int i){
		for(int j = 0; j < dims; j++){
			atomicAdd(&centroids[labels[i]*dims+j], dataset[i*dims+j]);
		}
	}
	
};

struct divide{
	int dims;
	float* centroids;
	int* labelcounts;

	divide(int _dims, float* _centroids, int* _labelcounts):
	dims(_dims), centroids(_centroids), labelcounts(_labelcounts){};

	__device__
	void operator()(int i){
		for(int j = 0; j < dims; j++){
			centroids[i*dims+j] = centroids[i*dims+j]/labelcounts[i];	
		}
	}
	
};

void kmeans(){
	thrust::device_vector<float> dataset(n*dims);
	thrust::device_vector<float> centroids(num_cluster*dims);
	thrust::device_vector<float> oldcentroids(num_cluster*dims);
	thrust::device_vector<int> labels(n);
	hostcentroids.resize(dims*num_cluster);
	hostlabels.resize(n);
    randomCentroids();
	dataset = hostdataset;
	centroids = hostcentroids;
    int iterations = 0;
    bool done = false;
	float totalduration = 0;
    while(!done){
		auto start = high_resolution_clock::now();
		//
		//
		// FIND NEAREST CENTROIDS 
		//
		//
		thrust::copy(centroids.begin(), centroids.end(), oldcentroids.begin());
		thrust::device_vector<float> oldcentroids= centroids;
        iterations++;
		thrust::fill(labels.begin(),labels.end(), 0);
		thrust::device_vector<int> temp(n);
		thrust::sequence(temp.begin(),temp.end());
		thrust::for_each(temp.begin(), temp.end(), findNearestCentroids(dims, num_cluster, thrust::raw_pointer_cast(dataset.data()), thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(labels.data())));
		//
		//
		// AVERAGE LABELED CENTROIDS
		//
		//
		thrust::device_vector<int> unsortedlabels(n);
		thrust::copy(labels.begin(), labels.end(), unsortedlabels.begin());
		thrust::sort(labels.begin(), labels.end());
		thrust::device_vector<int> ones(n);
		thrust::fill(ones.begin(),ones.end(), 1);
		thrust::device_vector<int> uniqueLabels(num_cluster);
		thrust::device_vector<int> labelsCount(num_cluster);
		thrust::reduce_by_key(labels.begin(), labels.end(), ones.begin(), uniqueLabels.begin(), labelsCount.begin());
		thrust::device_vector<int> ncount(n);
		thrust::sequence(ncount.begin(),ncount.end());
		thrust::fill(centroids.begin(), centroids.end(), 0);
		thrust::for_each(ncount.begin(), ncount.end(), newadd(dims, thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(dataset.data()), thrust::raw_pointer_cast(unsortedlabels.data())));
		thrust::device_vector<int> numclusters(num_cluster);
		thrust::sequence(numclusters.begin(),numclusters.end());
		thrust::for_each(numclusters.begin(),numclusters.end(), divide(dims, thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(labelsCount.data())));
		//
		//
		// CONVERGE CENTROIDS
		//
		//
		thrust::device_vector<int> convergedVector(num_cluster*dims);
		thrust::sequence(convergedVector.begin(), convergedVector.end());
		thrust::for_each(convergedVector.begin(), convergedVector.end(), converged(threshold, thrust::raw_pointer_cast(oldcentroids.data()),thrust::raw_pointer_cast(centroids.data()), thrust::raw_pointer_cast(convergedVector.data())));
		int sum = thrust::reduce(convergedVector.begin(), convergedVector.end(), (int) 0, thrust::plus<int>());
        done = (iterations == max_num_iter || sum == 0);
		auto stop = high_resolution_clock::now();
		float duration = duration_cast<milliseconds>(stop - start).count(); 
		totalduration += duration;
		hostcentroids=centroids;
		hostlabels=unsortedlabels;
    }
	float time_per_iter_in_ms = totalduration/iterations;
	auto iter_to_converge = iterations;
	printf("%d,%lf\n", iter_to_converge, time_per_iter_in_ms);
	if(flag){
		for(int i = 0; i < num_cluster; i++){
			printf("%d ", i);
			for(int j = 0; j < dims; j++){
				printf("%lf ", hostcentroids[i*dims +j]);
			}
			printf("\n");
		}
	}
	else{
		printf("clusters:");
		for (int p=0; p < n; p++)
			printf(" %d", hostlabels[p]);		
	}
}

int main(int argc, char* argv[]){
    string inputPath;
    for(int i = 1; i < argc; i++){
        string arg = string(argv[i]);
        if(arg == "-k"){
            num_cluster = stoi(argv[++i]);
        }
        if(arg == "-d"){
            dims = stoi(argv[++i]);
        }
        if(arg == "-i"){
            inputfilename = argv[++i]; 
        }
        if(arg == "-m"){
            max_num_iter = stoi(argv[++i]);
        }
        if(arg == "-t"){
            threshold = stod(argv[++i]);
        }
        if(arg == "-c"){
            flag = true;
        }
        if(arg == "-s"){
            seed = stoi(argv[++i]);
            kmeans_srand(seed);
        }
    }
    ifstream inputfile;
    inputfile.open(inputfilename);
    string stringn;
    getline(inputfile, stringn);
    n = stoi(stringn);
	hostdataset.resize(n*dims);
	int counter = 0;
    for(int i = 0; i < n; i++){
        string line;
        getline(inputfile,line);
        string value="";
        bool index = false;
        for(int j = 0; j < line.size(); j++){
            if(line[j] == ' '){
                if(index == false){
                    index = true;
                    value = "";
                    continue;
                }
				hostdataset[counter] = stod(value);
				counter++;
               value="";
            }
            else{
                value = value + line[j];
            }
        }
        if(value != ""){
			hostdataset[counter] = stod(value);
			counter++;
        }
    }
    kmeans();
}
