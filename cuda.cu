#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <vector>
#include <array>
#include <math.h>
#include <chrono>     
#include <iomanip>
using namespace std;
using namespace std::chrono; 

int n;
int num_cluster;
int dims;
string inputfilename;
int max_num_iter;
double threshold;
bool flag;
unsigned int seed = 8675309;

static unsigned long int nextstd = 1;
static unsigned long kmeans_rmax = 32767;
int kmeans_rand() {
    nextstd = nextstd * 1103515245 + 12345;
    return (unsigned int)(nextstd/65536) % (kmeans_rmax+1);
}
void kmeans_srand(unsigned int funcseed) {
    nextstd = funcseed;
}

void findNearestCentroids(int labels[], double dataset[], double centroids[]){
    for(int i = 0; i < n;i++){
		labels[i] = 0;
        int label = 0;
        double dist=0;
        double running = 0;
        for(int k = 0; k < dims; k++){
            running += pow(dataset[i*dims+k]-centroids[0*dims+k],2);
        }
        running = sqrt(running);
        dist = running;
        for(int j = 1; j < num_cluster; j++){
            double inner = 0;
            for(int k = 0; k < dims; k++){
                inner += pow(dataset[i*dims+k]-centroids[j*dims+k],2);
            }
            inner = sqrt(inner);
            if(inner < dist){
               label = j;
               dist = inner;
            }
        }
		labels[i] = label;
        //labels.push_back(label);
    }
}
void randomCentroids(double centroids[],double dataset[]){
	int count = 0;
    for (int i=0; i<num_cluster; i++){
        int index = kmeans_rand() % n;
        for (int j = 0; j < dims; j++){
			centroids[count] = dataset[index*dims+j];
			count++;
            //centroids.push_back(dataset[index*dims+j]);
        }
    }
}
void zeroArray(double centroids[]){
	for(int i = 0; i < num_cluster*dims;i++){
		centroids[i] = 0;
	}
}
void averageLabeledCentroids(double newcentroids[], double dataset[], int labels[]){
	double labelcounts[num_cluster]={0};
   for(int i = 0; i < n; i++){
       labelcounts[labels[i]] = labelcounts[labels[i]]+1;
       for(int j = 0; j < dims; j++){
        newcentroids[labels[i]*dims+j] += dataset[i*dims+j];
       }
   }

   for(int i = 0; i < num_cluster; i++){
       for(int j = 0; j < dims; j++){
           newcentroids[i*dims+j] = newcentroids[i*dims+j]/labelcounts[i];
       }
   }
}
bool converged(double oldCentroids[], double newCentroids[]){
	for(int i = 0; i < num_cluster*dims; i++){
		if(!(abs(oldCentroids[i] - newCentroids[i]) < threshold)){
			return false;
		}
	}
	return true;
}
void kmeans(double dataset[]){
	double centroids[num_cluster*dims]={0};
	int labels[n]={0};
    randomCentroids(centroids, dataset);
    int iterations = 0;
    bool done = false;
	double totalduration = 0;
    while(!done){
		auto start = high_resolution_clock::now();
		double oldCentroids[num_cluster*dims]={0};
        //vector<double> oldCentroids;
        for(int i = 0; i < num_cluster*dims; i++){
			oldCentroids[i] = centroids[i];
        }
        iterations++;
        findNearestCentroids(labels,dataset,centroids);
		zeroArray(centroids);
        averageLabeledCentroids(centroids,dataset,labels);
        done = (iterations == max_num_iter || converged(centroids, oldCentroids));
		auto stop = high_resolution_clock::now();
		double duration = duration_cast<microseconds>(stop - start).count(); 
		totalduration += duration;
    }
	double time_per_iter_in_ms = totalduration/iterations;
	auto iter_to_converge = iterations;
	printf("%d,%lf\n", iter_to_converge, time_per_iter_in_ms);
	if(flag){
		for(int i = 0; i < num_cluster; i++){
			printf("%d ", i);
			for(int j = 0; j < dims; j++){
				printf("%lf ", centroids[i*dims +j]);
			}
			printf("\n");
		}
	}
	else{
		printf("clusters:");
		for (int p=0; p < n; p++)
			printf(" %d", labels[p]);		
	}
}


int main(int argc, char* argv[]){
    string inputPath;
    for(int i = 1; i < argc; i++){
        string arg = string(argv[i]);
        if(arg == "-k"){
            num_cluster = stoi(argv[++i]);
        }
        if(arg == "-d"){
            dims = stoi(argv[++i]);
        }
        if(arg == "-i"){
            inputfilename = argv[++i]; 
        }
        if(arg == "-m"){
            max_num_iter = stoi(argv[++i]);
        }
        if(arg == "-t"){
            threshold = stod(argv[++i]);
        }
        if(arg == "-c"){
            flag = true;
        }
        if(arg == "-s"){
            seed = stoi(argv[++i]);
            kmeans_srand(seed);
        }
    }
    ifstream inputfile;
    inputfile.open(inputfilename);
    string stringn;
    getline(inputfile, stringn);
    n = stoi(stringn);
	double dataset[n*dims]={0};
	int count = 0;
    for(int i = 0; i < n; i++){
        string line;
        getline(inputfile,line);
        string value="";
        bool index = false;
        for(int j = 0; j < line.size(); j++){
            if(line[j] == ' '){
                if(index == false){
                    index = true;
                    value = "";
                    continue;
                }
				dataset[count] = stod(value);
               //dataset.push_back(stod(value)); 
			   count++;
               value="";
            }
            else{
                value = value + line[j];
            }
        }
        if(value != ""){
			dataset[count] = stod(value);
			count++;
            //dataset.push_back(stod(value));
        }
    }
    kmeans(dataset);
}
